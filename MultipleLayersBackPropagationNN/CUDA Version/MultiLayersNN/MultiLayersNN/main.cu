#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <cmath>
#include <vector>
#include <cstdlib>

using namespace std;

class Neuron
{
private:
    double value;
    int weightsNum;
    double derivationValue;
    vector<double> weights;
    void initWeghts()
    {
        for (size_t i = 0; i < weightsNum; i++)
        {
            // double t = rand() / RAND_MAX;
            double t = rand() % 10;
            weights.push_back(t);
        }
    }
public:
    Neuron(const int N);
    double valueOfNeuron() const { return value; };
    double getDerivationValue() const { return derivationValue; };
    void changeDerivationValue(double target);
    void updateValue(double newValue) { value = newValue; };
    vector<double> weightsOfNeuron() const { return this->weights; };
    double sqrtError(double target);
    double derivationOfError(double target);
    double sigmoid();
    double derivationOfSigmoid();
    double derivationOfTotalError(double target);
    ~Neuron();
};

Neuron::Neuron(const int N)
{
    value = 0.0;
    derivationValue = 0.0;
    weightsNum = N;
    this->initWeghts();
}

void Neuron::changeDerivationValue(double target)
{
    derivationValue = this->derivationOfTotalError(target);
}

double Neuron::sqrtError(double target)
{
    return pow(target - value, 2);
}

double Neuron::derivationOfError(double target)
{
    return value - target;
}

double Neuron::sigmoid()
{
    return 1 / (1 + exp(-this->value));
}

double Neuron::derivationOfSigmoid()
{
    double t = this->sigmoid();
    return t*(1 - t);
}

double Neuron::derivationOfTotalError(double target)
{
    double t = this->derivationOfError * this->derivationOfSigmoid();
    return t;
}

Neuron::~Neuron()
{}

class NeuronLayer
{
private:
    int nodes;
    vector<Neuron> layer;
public:
    NeuronLayer(const int M, const int N);
    int nodesOfLayer() const { return nodes; };
    vector<Neuron> layerOfNeuron() const { return this->layer; };
    ~NeuronLayer();
};

NeuronLayer::NeuronLayer(const int M, const int N)
{
    nodes = M;
    for (size_t i = 0; i < nodes; i++)
    {
        Neuron n(N);
        layer.push_back(n);
    }
}

NeuronLayer::~NeuronLayer()
{}

class NeuronNetwork
{
private:
    int layersNumber;
    vector<NeuronLayer> layers;
    vector<double> inputs;
    vector<double> targets;
    static double learningRate;
    static double errorPrecision;
public:
    NeuronNetwork(vector<double> & inputs, const int weightsNum, vector<int> & nodesNum, const int layersNum, vector<double> & targets);
    void forward();
    void backward();
    void updateDerivationOfNode();
    double updateWeight();
    void train();
    ~NeuronNetwork();
};

double NeuronNetwork::learningRate = 0.3;
double NeuronNetwork::errorPrecision = 1e-4;

NeuronNetwork::NeuronNetwork(vector<double> & inputs, const int weightsNum, vector<int> & nodesNum, const int layersNum, vector<double> & targets)
{
    inputs = inputs; targets = targets; layersNumber = layersNum;
    for (size_t i = 0; i < layersNumber; i++)
    {
        NeuronLayer layer(nodesNum[i], weightsNum);
        layers.push_back(layer);
    }
}

void NeuronNetwork::forward()
{
    vector<double> lastNodeValue;
    // update first layer
    for (size_t i = 0; i < layers[0].nodesOfLayer(); i++)
    {
        double t = 0.0;
        for (size_t j = 0; j < inputs.size(); j++)
        {
            t += inputs[j] * layers[0].layerOfNeuron()[i].weightsOfNeuron()[j];
        }
        layers[0].layerOfNeuron()[i].updateValue(t);
    }

    for (size_t i = 1; i < layersNumber; i++)
    {
        for (size_t j = 0; j < layers[i].nodesOfLayer(); j++)
        {
            double t = 0.0;
            for (size_t k = 0; k < layers[i].layerOfNeuron()[j].weightsOfNeuron().size(); k++)
            {
                t += layers[i].layerOfNeuron()[j].weightsOfNeuron()[k] * layers[i - 1].layerOfNeuron()[k].valueOfNeuron();
            }
            layers[i].layerOfNeuron()[j].updateValue(t);
        }
    }
}

void NeuronNetwork::updateDerivationOfNode()
{
    // update output layer first
    for (size_t i = 0; i < layers[layersNumber - 1].nodesOfLayer(); i++)
    {
        double t = 0.0;
        t = (layers[layersNumber - 1].layerOfNeuron()[i].derivationOfTotalError(targets[i])) * layers[layersNumber - 1].layerOfNeuron()[i].derivationOfSigmoid();
        layers[layersNumber - 1].layerOfNeuron()[i].changeDerivationValue(t);
    }
    // update hidden layers
    for (size_t i = layersNumber - 2; i >= 0; i--)
    {
        for (size_t j = 0; j < layers[i].nodesOfLayer(); i++)
        {
            //layers[i].layerOfNeuron()[j]
            double t = 0.0;
            for (size_t k = 0; k < layers[i + 1].layerOfNeuron().size(); k++)
            {
                t += layers[i + 1].layerOfNeuron()[k].getDerivationValue() * layers[i + 1].layerOfNeuron()[k].weightsOfNeuron()[j];
            }
            layers[i].layerOfNeuron()[j].changeDerivationValue(t);
        }
    }
}

double NeuronNetwork::updateWeight()
{
    // update output layer firstly
    for (size_t i = 0; i < layers[layersNumber-1].nodesOfLayer(); i++)
    {
        for (size_t j = 0; j < layers[layersNumber - 1].layerOfNeuron()[i].weightsOfNeuron().size(); j++)
        {
            double t = layers[layersNumber - 1].layerOfNeuron()[i].derivationOfTotalError(targets[i])*layers[layersNumber - 2].layerOfNeuron()[j].valueOfNeuron();
            t = layers[layersNumber - 1].layerOfNeuron()[i].weightsOfNeuron()[j] - learningRate*t;
        }
    }
    // update hidden layers weights
    for (size_t i = layersNumber - 2; i >= 0; i--)
    {

    }
}

NeuronNetwork::~NeuronNetwork()
{}

int main()
{
    const int N = 3;
    Neuron n(N);
    cout << n.valueOfNeuron() << endl;
    for (size_t i = 0; i < N; i++)
    {
        cout << n.weightsOfNeuron()[i] << endl;
    }
    system("pause");
    return 0;
}